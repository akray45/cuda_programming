#include <iostream>
#include <stdlib.h>
#include <string>
#include "hip/hip_runtime.h"
using namespace std;

__global__ void hello_cuda(){
    printf("hello threadIdx %d, blockIdx %d \n",threadIdx.x, blockIdx.x);

}

int main(){
    hello_cuda<<<1, 2>>>();
    hipDeviceSynchronize();
    return 0;
}