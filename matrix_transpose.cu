#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <ctime>
#include <cassert>
#include <vector>
#include "hip/hip_runtime.h"
#include "ERROR.h"

using namespace std;

__global__ void transpose(int *a, int *b, int size){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    b[i*size + j] = b[j*size + i];
}

void generate_random(vector<vector<int>>&a, int size){
    srand(time(NULL));
    for(int i = 0; i<size; i++)
    {   vector<int> temp;
        for (int j = 0; j<size; j++){
        temp.push_back(rand() %size +1);
    }
    a.push_back(temp);
}


}
void verify_result(vector<vector<int>> &a, vector<vector<int>> &b){
    for(int i=0; i<a.size();i++){
        for(int j = 0; j<a.size(); j++){
            assert(b[i][j] == a[j][i]);
        }
    }
    printf("result verified.....\n");
}



void device(vector<vector<int>>&a, vector<vector<int>> &b, int size){
    int *d_a, *d_b;
    int total_size = size*sizeof(int)*size;
    HANDLER_ERROR_ERR(hipMalloc( &d_a, total_size ));
    HANDLER_ERROR_ERR(hipMalloc( &d_b, total_size ));

    hipMemcpy( d_a,a.data() ,total_size , hipMemcpyHostToDevice);
    hipMemcpy( d_b,b.data() ,total_size , hipMemcpyHostToDevice);
    dim3 GridBlocks(4, 4);
    dim3 ThreadsBlocks(16, 16);

    //kernel launch
    transpose <<< GridBlocks,ThreadsBlocks >>>(d_a, d_b, size);
    hipDeviceSynchronize();

    hipMemcpy( b.data(),d_b ,total_size , hipMemcpyDeviceToHost);
    HANDLER_ERROR_ERR(hipFree( d_a));
    HANDLER_ERROR_ERR(hipFree( d_b));


}

void host(){
    int size = 1000;
    vector<vector<int>> a;
    vector<vector<int>> b;
    device(a, b, size);
    verify_result(a, b);


}

int main(){
    host();
    return 0;
}