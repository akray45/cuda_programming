// #include <iostream>
// #include <stdlib.h>
// #include <ctime>
#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
using namespace std;
#include <vector>
#include <cassert>

__global__ void mat_add(int *a, int *b, int *c, int count){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < count)
            c[tid] = a[tid] + b[tid];


}

void print_array(vector<int> &a, int count){
    for(int i = 0; i<count; i++){
        cout << "element at index "<<i<<" :- "<<a[i]<<endl;
    }
}

void generate_random(vector <int> &a, int count){
    // srand(time(NULL));

    for (int i =0; i<count; i++){
        a.push_back(rand() %100 + 1);
    }
}

void verify_result(vector <int> &a, vector <int> &b, vector <int> &c){
    for(int i = 0; i<a.size(); i++){
        assert(c[i] == a[i]+b[i]);
    }
}


int main(){
    int count = 200;
    vector<int> a;
    vector<int> b;
    vector<int> c(count);
    
    int size = sizeof(int) *count;

    int NUM_THREADS = 256;
    int NUM_BLOCKS = 1;

    generate_random(a, count);
    generate_random(b, count);

    print_array(a, 6);
    print_array(b, 6);


    int *d_a, *d_b, *d_c;
    hipMalloc( &d_a,size );
    hipMalloc( &d_b, size );
    hipMalloc( &d_c, size );

    hipMemcpy( d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy( d_b, b.data(),size , hipMemcpyHostToDevice);

    //kernel call
    mat_add<<< NUM_BLOCKS, NUM_THREADS >>>(d_a, d_b, d_c, count);

    hipDeviceSynchronize();
    // writing data on cpu from gpu 
    hipMemcpy( c.data(), d_c ,size , hipMemcpyDeviceToHost);



    verify_result(a, b, c);
    print_array(c, 6);

    hipFree( d_a);
    hipFree( d_b);
    hipFree( d_c);

    return 0;







}